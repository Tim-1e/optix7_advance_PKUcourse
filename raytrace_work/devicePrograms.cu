#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2018-2019 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include <optix_device.h>
#include <hip/hip_runtime.h>
#include <vector>
#include "BDPT_function.h"
#include "config.h"
using namespace osc;

namespace osc
{

    /*! launch parameters in constant memory, filled in by optix upon
        optixLaunch (this gets filled in from the buffer we pass to
        optixLaunch) */
    extern "C" __constant__ LaunchParams optixLaunchParams;

    //------------------------------------------------------------------------------
    // closest hit and anyhit programs for radiance-type rays.
    //
    // Note eventually we will have to create one pair of those for each
    // ray type and each geometry type we want to render; but this
    // simple example doesn't use any actual geometries yet, so we only
    // create a single, dummy, set of them (we do have to have at least
    // one group of them to set up the SBT)
    //------------------------------------------------------------------------------

    extern "C" __global__ void __closesthit__shadow()
    {
        const TriangleMeshSBTData& sbtData
            = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();
        vec2i& dir_hit = *getPRD<vec2i>();
        if (dir_hit.x == sbtData.ID && dir_hit.y == optixGetPrimitiveIndex()) {
            dir_hit.x = -1;
        }
    }

    extern "C" __global__ void __closesthit__radiance()
    {
        const TriangleMeshSBTData& sbtData
            = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();
        PRD& prd = *getPRD<PRD>();
        if (prd.depth >= Maxdepth) {
            prd.end = 1;
            return;
        }
        if (sbtData.emissive_) {
            prd.end = 1;
            return;
        }
        // ------------------------------------------------------------------
        // gather some basic hit information
        // ------------------------------------------------------------------
        const int primID = optixGetPrimitiveIndex();
        const vec3i index = sbtData.index[primID];
        const float u = optixGetTriangleBarycentrics().x;
        const float v = optixGetTriangleBarycentrics().y;

        // ------------------------------------------------------------------
        // compute normal, using either shading normal (if avail), or
        // geometry normal (fallback)
        // ------------------------------------------------------------------
        const vec3f &A = sbtData.vertex[index.x];
        const vec3f &B = sbtData.vertex[index.y];
        const vec3f &C = sbtData.vertex[index.z];
        vec3f Ng = cross(B - A, C - A);
        vec3f Ns = (sbtData.normal)
                       ? ((1.f - u - v) * sbtData.normal[index.x] + u * sbtData.normal[index.y] + v * sbtData.normal[index.z])
                       : Ng;

        // ------------------------------------------------------------------
        // face-forward and normalize normals
        // ------------------------------------------------------------------
        const vec3f rayDir = optixGetWorldRayDirection();
        
        if (dot(rayDir, Ng) > 0.f) Ng = -Ng;
        Ng = normalize(Ng);

        if (dot(Ng, Ns) < 0.f)
            Ns -= 2.f * dot(Ng, Ns) * Ng;
        Ns = normalize(Ns);

        // ------------------------------------------------------------------
        // compute diffuse material color, including diffuse texture, if
        // available
        // ------------------------------------------------------------------
        vec3f diffuseColor = sbtData.color;
        if (sbtData.hasTexture && sbtData.texcoord)
        {
            const vec2f tc = (1.f - u - v) * sbtData.texcoord[index.x] + u * sbtData.texcoord[index.y] + v * sbtData.texcoord[index.z];

            vec4f fromTexture = tex2D<float4>(sbtData.texture, tc.x, tc.y);
            diffuseColor *= (vec3f)fromTexture;
        }

        vec3f specColor = 0.0f;
        if (sbtData.hasSpecTexture && sbtData.texcoord)
        {
            const vec2f tc = (1.f - u - v) * sbtData.texcoord[index.x] + u * sbtData.texcoord[index.y] + v * sbtData.texcoord[index.z];
            vec4f fromTexture = tex2D<float4>(sbtData.spectexture, tc.x, tc.y);
            specColor = (vec3f)fromTexture;
        }

        //const float alpha = sbtData.alpha_;
        //const float d = sbtData.d;


        // ------------------------------------------------------------------
        // compute shadow
        // ------------------------------------------------------------------
        const vec3f surfPos = (1.f - u - v) * sbtData.vertex[index.x] + u * sbtData.vertex[index.y] + v * sbtData.vertex[index.z];

        float diffuse_max = max(max(diffuseColor[0], diffuseColor[1]), diffuseColor[2]);
        
        const float RR = 0.8f;//clamp(diffuse_max,0.3f,0.9f);//����˹���̶�
        if (prd.random() > RR) {
            prd.end = 1;
            return;
        }

        vec3f mont_dir;//�ⷽ��
        M_extansion mext;
        mext.diffuseColor = diffuseColor;
        mext.specColor = specColor;//��������

        //Pass ���µ����path

        //std::printf("initing prd\n");
        //std::printf("length:%d,depth:%d\n",prd.path->length,prd.depth);
        prd.path->vertexs[prd.depth].init(surfPos, Ns, (TriangleMeshSBTData*)optixGetSbtDataPointer(), mext,primID);
        //std::printf("vertexs init finished\n");
        prd.path->length = prd.depth + 1;
        //ȡ��·������
        mont_dir = Sample_adjust(sbtData, Ns, rayDir,prd);
        prd.depth = prd.depth + 1;
        prd.normal = Ng;
        prd.sourcePos = surfPos;
        prd.direction = mont_dir;
        return;
    }

    extern "C" __global__ void __anyhit__radiance()
    { /*! for this simple example, this will remain empty */
    }

    extern "C" __global__ void __anyhit__shadow()
    { /*! not going to be used */
    }

    //------------------------------------------------------------------------------
    // miss program that gets called for any ray that did not have a
    // valid intersection
    //
    // as with the anyhit/closest hit programs, in this example we only
    // need to have _some_ dummy function to set up a valid SBT
    // ------------------------------------------------------------------------------

    extern "C" __global__ void __miss__radiance()
    {
        PRD& prd = *getPRD<PRD>();
        prd.end = 1;
        return;
    }

    extern "C" __global__ void __miss__shadow()
    {
    }

    //------------------------------------------------------------------------------
    // ray gen program - the actual rendering happens in here
    //------------------------------------------------------------------------------
    extern "C" __global__ void __raygen__renderFrame()
    {
        //BDPTPath path;
        //path.vertexs[0].init(vec3f(0));
        //std::printf("%f\n", path.vertexs[0].pdf);
        
        //const float color_max_avilable = 1.f;
        // compute a test pattern based on pixel ID
        const int ix = optixGetLaunchIndex().x;
        const int iy = optixGetLaunchIndex().y;
        const auto &camera = optixLaunchParams.camera;

        PRD prd;
        prd.random.init(ix + optixLaunchParams.frame.size.x * iy,
                        optixLaunchParams.frame.frameID);
        // the values we store the PRD pointer in:
        uint32_t u0, u1;
        packPointer(&prd, u0, u1);

        int numPixelSamples = optixLaunchParams.numPixelSamples;

        vec3f pixelColor = 0.f;
        //vec3f pixelNormal = 0.f;
        //vec3f pixelAlbedo = 0.f;

        for (int sampleID = 0; sampleID < numPixelSamples; sampleID++)
        {
            vec2f screen(vec2f(ix + prd.random(), iy + prd.random())
                / vec2f(optixLaunchParams.frame.size));


            // generate ray direction
            vec3f rayDir = normalize(camera.direction + (screen.x - 0.5f) * camera.horizontal + (screen.y - 0.5f) * camera.vertical);
            BDPTPath eye_path,light_path;
            //std::printf("pdf %f\n", eye_path.vertexs[0].pdf);
            //Begin the eye path build

            eye_path.vertexs[0].init(camera.position);
            eye_path.vertexs[0].normal = camera.direction;
            eye_path.length = 1;
            prd.depth = 1;
            prd.path=&eye_path;
            prd.end = 0;
            optixTrace(optixLaunchParams.traversable,
                camera.position,
                rayDir,
                0.f,    // tmin
                1e20f,  // tmax
                0.0f,   // rayTime
                OptixVisibilityMask(255),
                OPTIX_RAY_FLAG_DISABLE_ANYHIT,//OPTIX_RAY_FLAG_NONE,
                RADIANCE_RAY_TYPE,            // SBT offset
                RAY_TYPE_COUNT,               // SBT stride
                RADIANCE_RAY_TYPE,            // missSBTIndex 
                u0, u1);
            while (!prd.end)
            {
                optixTrace(optixLaunchParams.traversable,
                    prd.sourcePos + 1e-3f * prd.normal,
                    prd.direction,
                    0.f,    // tmin
                    1e20f,  // tmax
                    0.0f,   // rayTime
                    OptixVisibilityMask(255),
                    OPTIX_RAY_FLAG_DISABLE_ANYHIT,
                    RADIANCE_RAY_TYPE,            // SBT offset`
                    RAY_TYPE_COUNT,               // SBT stride
                    RADIANCE_RAY_TYPE,            // missSBTIndex 
                    u0, u1);
            }
            
            //Begin the light path build
            int num = optixLaunchParams.Lights_num;
            LightParams* Lp = &optixLaunchParams.All_Lights[int(num * prd.random())];
            LightSample Light_point;
            Lp->sample(Light_point, prd.random);

            light_path.length = 1;
            light_path.vertexs[0].pdf = Light_point.pdf;
            TriangleMeshSBTData mat;
            mat.ID = Light_point.id;
            mat.emission = Lp->emission;
            M_extansion ext;
            light_path.vertexs[0].init(Light_point.position, Light_point.normal,&mat,ext,Light_point.meshID);

            prd.depth = 1;
            prd.path = &light_path;
            prd.end = 0;
            rayDir = Lp->UniformSampleDir(Light_point.position,Light_point.normal,prd.random);
            optixTrace(optixLaunchParams.traversable,
                Light_point.position,
                rayDir,
                0.f,    // tmin
                1e20f,  // tmax
                0.0f,   // rayTime
                OptixVisibilityMask(255),
                OPTIX_RAY_FLAG_DISABLE_ANYHIT,//OPTIX_RAY_FLAG_NONE,
                RADIANCE_RAY_TYPE,            // SBT offset
                RAY_TYPE_COUNT,               // SBT stride
                RADIANCE_RAY_TYPE,            // missSBTIndex 
                u0, u1);
            while (!prd.end)
            {
                optixTrace(optixLaunchParams.traversable,
                    prd.sourcePos + 1e-3f * prd.normal,
                    prd.direction,
                    0.f,    // tmin
                    1e20f,  // tmax
                    0.0f,   // rayTime
                    OptixVisibilityMask(255),
                    OPTIX_RAY_FLAG_DISABLE_ANYHIT,
                    RADIANCE_RAY_TYPE,            // SBT offset`
                    RAY_TYPE_COUNT,               // SBT stride
                    RADIANCE_RAY_TYPE,            // missSBTIndex 
                    u0, u1);
            }

            //std::printf("l_pdf %f\n", light_path.vertexs[0].pdf);
            for (int eye_length = 2; eye_length <= eye_path.length; eye_length++)
            {
                for (int light_length = 1; light_length <= 1; light_length++)
                {
                    //�ɼ����ж�
                    //std::printf("c_pdf %f\n", eye_path.vertexs[0].pdf);
                    vec3f eyeLastPoint = eye_path.vertexs[eye_length - 1].position;
                    vec3f Ng = eye_path.vertexs[eye_length - 1].normal;
                    vec3f lightLastPoint = light_path.vertexs[light_length - 1].position;
                    vec3f lightDir = normalize(lightLastPoint - eyeLastPoint);
                    //std::printf("initing connection\n");
                    vec2i dir_hit = vec2i(light_path.vertexs[light_length - 1].mat->ID, light_path.vertexs[light_length - 1].MeshID);
                    //std::printf("tracing\n");
                    packPointer(&dir_hit, u0, u1);
                    optixTrace(optixLaunchParams.traversable,
                        eyeLastPoint + 1e-3f * Ng,
                        lightDir,
                        0.f,    // tmin
                        1e20f,  // tmax
                        0.0f,   // rayTime
                        OptixVisibilityMask(255),
                        OPTIX_RAY_FLAG_DISABLE_ANYHIT,
                        SHADOW_RAY_TYPE,            // SBT offset
                        RAY_TYPE_COUNT,               // SBT stride
                        SHADOW_RAY_TYPE,            // missSBTIndex 
                        u0, u1);
                    //printf("we try?\n");
                    if (dir_hit.x == -1)
                    {
                        //std::printf("color %f\n", pixelColor.r);
                        BDPTPath connect_path;
                        Connect_two_path(eye_path, light_path, connect_path, eye_length, light_length);
                        pixelColor += evalPath(connect_path);
                        //std::printf("color %f\n", evalPath(connect_path).r);
                        
                    }
                }
            }
            
        }

        vec4f rgba(pixelColor / numPixelSamples, 1.f);
        //vec4f albedo(pixelAlbedo / numPixelSamples, 1.f);
        //vec4f normal(pixelNormal / numPixelSamples, 1.f);
        
        // and write/accumulate to frame buffer ...
        const uint32_t fbIndex = ix + iy * optixLaunchParams.frame.size.x;
        if (optixLaunchParams.frame.frameID > 0)
        {
            rgba += float(optixLaunchParams.frame.frameID) * vec4f(optixLaunchParams.frame.colorBuffer[fbIndex]);
            rgba /= (optixLaunchParams.frame.frameID + 1.f);
        }
        optixLaunchParams.frame.colorBuffer[fbIndex] = (float4)rgba;
        //optixLaunchParams.frame.albedoBuffer[fbIndex] = (float4)albedo;
        //optixLaunchParams.frame.normalBuffer[fbIndex] = (float4)normal;
    }

} // ::osc
