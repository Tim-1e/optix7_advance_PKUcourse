#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2018-2019 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include <optix_device.h>
#include <hip/hip_runtime.h>
#include <vector>
#include "config.h"
#include "LaunchParams.h"
#include "tool_function.h"

using namespace osc;

namespace osc
{

    /*! launch parameters in constant memory, filled in by optix upon
        optixLaunch (this gets filled in from the buffer we pass to
        optixLaunch) */
    extern "C" __constant__ LaunchParams optixLaunchParams;
    //------------------------------------------------------------------------------
    // closest hit and anyhit programs for radiance-type rays.
    //
    // Note eventually we will have to create one pair of those for each
    // ray type and each geometry type we want to render; but this
    // simple example doesn't use any actual geometries yet, so we only
    // create a single, dummy, set of them (we do have to have at least
    // one group of them to set up the SBT)
    //------------------------------------------------------------------------------

    extern "C" __global__ void __closesthit__shadow()
    {
        const TriangleMeshSBTData& sbtData
            = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();
        int& light_hit = *getPRD<int>();

        if (light_hit == sbtData.ID) {
            light_hit = -1;
        }
    }

    extern "C" __global__ void __closesthit__radiance()
    {
        const TriangleMeshSBTData& sbtData
            = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();
        PRD& prd = *getPRD<PRD>();
        if (prd.depth >= MAX_DEPTH) {
            prd.pixelColor = vec3f(0.0f);
            prd.end = 1;
            return;
        }
        if (sbtData.emissive_) {
                int MeshId = sbtData.ID;
                int PrimId = optixGetPrimitiveIndex();
                int num = optixLaunchParams.Lights_num;
                vec3f light_pdf;
                switch (MY_MODE) {
                case MY_MIS:
                    for (int i = 0; i < num; i++)
                    {
                        if (optixLaunchParams.All_Lights[i].id == MeshId)
                        {
                            LightParams* hit_light = &optixLaunchParams.All_Lights[i];
                            LightSample hit_point;
                            hit_light->sample(hit_point, prd.random, PrimId);
                            light_pdf = hit_point.Pdf_Light(prd.sourcePos, prd.nextPosition);
                            break;
                        }
                    }
                    prd.pixelColor = sbtData.emission * prd.throughout/(prd.weight+ light_pdf * num);
                    break;
                case MY_BRDF:
                    prd.pixelColor = sbtData.emission * prd.throughout ;
                    break;
                case MY_NEE:
                    prd.pixelColor = vec3f(0);
                }
                prd.end = 1;
                return;
        }            
        prd.throughout /= prd.weight;//�ǹ�Դ�����뼴��

        // ------------------------------------------------------------------
        // gather some basic hit information
        // ------------------------------------------------------------------
        const int primID = optixGetPrimitiveIndex();
        const vec3i index = sbtData.index[primID];
        const float u = optixGetTriangleBarycentrics().x;
        const float v = optixGetTriangleBarycentrics().y;

        // ------------------------------------------------------------------
        // compute normal, using either shading normal (if avail), or
        // geometry normal (fallback)
        // ------------------------------------------------------------------
        const vec3f &A = sbtData.vertex[index.x];
        const vec3f &B = sbtData.vertex[index.y];
        const vec3f &C = sbtData.vertex[index.z];
        vec3f Ng = cross(B - A, C - A);
        vec3f Ns = (sbtData.normal)
                       ? ((1.f - u - v) * sbtData.normal[index.x] + u * sbtData.normal[index.y] + v * sbtData.normal[index.z])
                       : Ng;

        // ------------------------------------------------------------------
        // face-forward and normalize normals
        // ------------------------------------------------------------------
        const vec3f rayDir = optixGetWorldRayDirection();
        
        if (dot(rayDir, Ng) > 0.f) Ng = -Ng;
        Ng = normalize(Ng);

        if (dot(Ng, Ns) < 0.f)
            Ns -= 2.f * dot(Ng, Ns) * Ng;
        Ns = normalize(Ns);

        // ------------------------------------------------------------------
        // compute diffuse material color, including diffuse texture, if
        // available
        // ------------------------------------------------------------------
        vec3f diffuseColor = sbtData.color;
        if (sbtData.hasTexture && sbtData.texcoord)
        {
            const vec2f tc = (1.f - u - v) * sbtData.texcoord[index.x] + u * sbtData.texcoord[index.y] + v * sbtData.texcoord[index.z];

            vec4f fromTexture = tex2D<float4>(sbtData.texture, tc.x, tc.y);
            diffuseColor *= (vec3f)fromTexture;
        }

        vec3f specColor = 0.0f;
        if (sbtData.hasSpecTexture && sbtData.texcoord)
        {
            const vec2f tc = (1.f - u - v) * sbtData.texcoord[index.x] + u * sbtData.texcoord[index.y] + v * sbtData.texcoord[index.z];
            vec4f fromTexture = tex2D<float4>(sbtData.spectexture, tc.x, tc.y);
            specColor = (vec3f)fromTexture;
        }

        const float alpha = sbtData.alpha_;
        const float d = sbtData.d;

        const vec3f surfPos = (1.f - u - v) * sbtData.vertex[index.x] + u * sbtData.vertex[index.y] + v * sbtData.vertex[index.z];

        float diffuse_max = max(max(diffuseColor[0], diffuseColor[1]), diffuseColor[2]);
     
        // ------------------------------------------------------------------
        //Begin of the true brdf
        // ------------------------------------------------------------------

        uint32_t u0, u1;
        
        vec3f new_dir;//�ⷽ��
        vec3f weight = 1.0f;//Ȩ��

        M_extansion mext;
        mext.diffuseColor = diffuseColor;
        mext.specColor = specColor;//��������
        //ֱ�ӹ�
        int lightNum = optixLaunchParams.Lights_num;
        LightParams *LP = &optixLaunchParams.All_Lights[int(lightNum * prd.random())];
        LightSample LS;

        LP->sample(LS, prd.random,int(LP->num*prd.random()));

        int light_hit = LP->id;

        packPointer(&light_hit, u0, u1);
        vec3f lightDir = normalize(LS.position - surfPos);
        optixTrace(optixLaunchParams.traversable,
            surfPos + 1e-3f * Ng,
            lightDir,
            0.f,    // tmin
            1e20f,  // tmax
            0.0f,   // rayTime
            OptixVisibilityMask(255),
            OPTIX_RAY_FLAG_DISABLE_ANYHIT, 
            SHADOW_RAY_TYPE,            // SBT offset
            RAY_TYPE_COUNT,               // SBT stride
            SHADOW_RAY_TYPE,            // missSBTIndex 
            u0, u1);

        if (light_hit == -1) {
            float dis = length(LS.position - surfPos);
            weight *= lightNum;
            weight *= Eval(sbtData, Ns, rayDir, lightDir, mext);
            vec3f Dir_color_contri = prd.throughout * weight  * LS.emission ;
            float Pdf_nee = LS.pdf * dis * dis / dot(LS.normal, -lightDir);
            switch (MY_MODE)
            {
            case MY_BRDF:
                break;
            case MY_NEE:
                prd.pixelColor = Dir_color_contri / (Pdf_nee);
                break;
            case MY_MIS:
                prd.pixelColor = Dir_color_contri / (Pdf_nee + Pdf_brdf(sbtData, Ns, rayDir, lightDir));
                break;
            }
        }
            
        const float RR = clamp(diffuse_max, 0.3f, 0.9f);//����˹���̶�

        new_dir = SampleNewRay(sbtData, Ns, rayDir, prd);
        weight = Eval(sbtData, Ns, rayDir, new_dir, mext);
        prd.depth = prd.depth + 1;
        prd.throughout = prd.throughout * weight / RR;
        prd.sourcePos = surfPos;
        prd.nextPosition = new_dir;


        prd.weight = Pdf_brdf(sbtData, Ns, rayDir, new_dir);
        prd.throughout = min(prd.throughout, vec3f(1e3f));
        prd.pixelNormal = Ns;
        prd.pixelAlbedo = diffuseColor;
        prd.pixelColor = max(prd.pixelColor,vec3f(0.f));

        if (prd.random() > RR) {
            prd.end = 1;
            return;
        }
    }

    extern "C" __global__ void __anyhit__radiance()
    { /*! for this simple example, this will remain empty */
    }

    extern "C" __global__ void __anyhit__shadow()
    { /*! not going to be used */
    }

    //------------------------------------------------------------------------------
    // miss program that gets called for any ray that did not have a
    // valid intersection
    //
    // as with the anyhit/closest hit programs, in this example we only
    // need to have _some_ dummy function to set up a valid SBT
    // ------------------------------------------------------------------------------

    extern "C" __global__ void __miss__radiance()
    {
        PRD &prd = *getPRD<PRD>();
        prd.pixelColor = 0.f;
        prd.end = 1;
    }

    extern "C" __global__ void __miss__shadow()
    {

    }

    //------------------------------------------------------------------------------
    // ray gen program - the actual rendering happens in here
    //------------------------------------------------------------------------------
    extern "C" __global__ void __raygen__renderFrame()
    {
        const float color_max_avilable = 1.f;
        const int ix = optixGetLaunchIndex().x;
        const int iy = optixGetLaunchIndex().y;
        const auto &camera = optixLaunchParams.camera;

        PRD prd;
        prd.random.init(ix + optixLaunchParams.frame.size.x * iy,
                        optixLaunchParams.frame.frameID);
        // the values we store the PRD pointer in:
        uint32_t u0, u1;
        packPointer(&prd, u0, u1);

        int numPixelSamples = optixLaunchParams.numPixelSamples;

        vec3f pixelColor = 0.f;
        vec3f pixelNormal = 0.f;
        vec3f pixelAlbedo = 0.f;
        for (int sampleID = 0; sampleID < numPixelSamples; sampleID++)
        {
            // normalized screen plane position, in [0,1]^2

            // iw: note for denoising that's not actually correct - if we
            // assume that the camera should only(!) cover the denoised
            // screen then the actual screen plane we shuld be using during
            // rendreing is slightly larger than [0,1]^2
            vec2f screen(vec2f(ix + prd.random(), iy + prd.random())
                / vec2f(optixLaunchParams.frame.size));
 
            // generate ray direction
            vec3f rayDir = normalize(camera.direction + (screen.x - 0.5f) * camera.horizontal + (screen.y - 0.5f) * camera.vertical);
            
            prd.pixelColor = vec3f(0.f);
            prd.pixelAlbedo = vec3f(0.f);
            prd.pixelNormal = vec3f(0.f);
            prd.depth = 0;
            prd.throughout = vec3f(1.f);
            prd.sourcePos = camera.position;
            prd.weight = vec3f(1.f);
            prd.end = 0;
            optixTrace(optixLaunchParams.traversable,
                camera.position,
                rayDir,
                0.f,    // tmin
                1e20f,  // tmax
                0.0f,   // rayTime
                OptixVisibilityMask(255),
                OPTIX_RAY_FLAG_DISABLE_ANYHIT,//OPTIX_RAY_FLAG_NONE,
                RADIANCE_RAY_TYPE,            // SBT offset
                RAY_TYPE_COUNT,               // SBT stride
                RADIANCE_RAY_TYPE,            // missSBTIndex 
                u0, u1);
            pixelColor += prd.pixelColor;
            pixelNormal += prd.pixelNormal;
            pixelAlbedo += prd.pixelAlbedo;

            while (!prd.end)
            {
                //��ӹ�
                optixTrace(optixLaunchParams.traversable,
                    prd.sourcePos + 1e-3f * prd.pixelNormal,
                    prd.nextPosition,
                    0.f,    // tmin
                    1e20f,  // tmax
                    0.0f,   // rayTime
                    OptixVisibilityMask(255),
                    OPTIX_RAY_FLAG_DISABLE_ANYHIT,
                    RADIANCE_RAY_TYPE,            // SBT offset`
                    RAY_TYPE_COUNT,               // SBT stride
                    RADIANCE_RAY_TYPE,            // missSBTIndex 
                    u0, u1);
                    pixelColor += max(prd.pixelColor, vec3f(0.f));
            }
            //printf("End!!");
        }

        vec4f rgba(pixelColor / numPixelSamples, 1.f);
        vec4f albedo(pixelAlbedo / numPixelSamples, 1.f);
        vec4f normal(pixelNormal / numPixelSamples, 1.f);

        // and write/accumulate to frame buffer ...
        const uint32_t fbIndex = ix + iy * optixLaunchParams.frame.size.x;
        if (optixLaunchParams.frame.frameID > 0)
        {
            rgba += float(optixLaunchParams.frame.frameID) * vec4f(optixLaunchParams.frame.colorBuffer[fbIndex]);
            rgba /= (optixLaunchParams.frame.frameID + 1.f);
        }
        optixLaunchParams.frame.colorBuffer[fbIndex] = (float4)rgba;
        optixLaunchParams.frame.albedoBuffer[fbIndex] = (float4)albedo;
        optixLaunchParams.frame.normalBuffer[fbIndex] = (float4)normal;
        //printf("we got rgba as %f   %f    %f    %f\n", rgba.x, rgba.y, rgba.z, rgba.w);
    }

} // ::osc
