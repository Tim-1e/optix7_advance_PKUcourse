#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2018-2019 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include <optix_device.h>
#include <hip/hip_runtime.h>

#include "LaunchParams.h"
#include "gdt/random/random.h"

using namespace osc;

#define NUM_LIGHT_SAMPLES 2

namespace osc {

    typedef gdt::LCG<16> Random;

    /*! launch parameters in constant memory, filled in by optix upon
        optixLaunch (this gets filled in from the buffer we pass to
        optixLaunch) */
    extern "C" __constant__ LaunchParams optixLaunchParams;

    /*! per-ray data now captures random number generator, so programs
        can access RNG state */
    struct PRD {
        Random random;
        int depth;
        vec3f  pixelColor;
        vec3f  pixelNormal;
        vec3f  pixelAlbedo;
        float refraction_index;//��ǰ��������ڿ�����������
    };

    static __forceinline__ __device__
        void* unpackPointer(uint32_t i0, uint32_t i1)
    {
        const uint64_t uptr = static_cast<uint64_t>(i0) << 32 | i1;
        void* ptr = reinterpret_cast<void*>(uptr);
        return ptr;
    }

    static __forceinline__ __device__
        void  packPointer(void* ptr, uint32_t& i0, uint32_t& i1)
    {
        const uint64_t uptr = reinterpret_cast<uint64_t>(ptr);
        i0 = uptr >> 32;
        i1 = uptr & 0x00000000ffffffff;
    }

    template<typename T>
    static __forceinline__ __device__ T* getPRD()
    {
        const uint32_t u0 = optixGetPayload_0();
        const uint32_t u1 = optixGetPayload_1();
        return reinterpret_cast<T*>(unpackPointer(u0, u1));
    }

    /*
*  Calculates refraction direction
*  lDir   : refraction vector
*  i   : incident vector
*  n   : surface normal
*  ior : index of refraction ( n2 / n1 )
*  returns false in case of total internal reflection, in that case lDir is
*  initialized to (0,0,0).
*/
    static __forceinline__  __device__
        bool refract(vec3f& lDir, vec3f const& i, vec3f const& n, const float ior)
    {
        vec3f nn = n;
        float negNdotV = dot(i, nn);
        float eta;

        if (negNdotV > 0.0f)
        {
            eta = ior;
            nn = -n;
            negNdotV = -negNdotV;
        }
        else
        {
            eta = 1.f / ior;
        }

        const float k = 1.f - eta * eta * (1.f - negNdotV * negNdotV);

        if (k < 0.0f)
        {
            // Initialize this value, so that lDir always leaves this function initialized.
            lDir = vec3f(0.f);
            return false;
        }
        else
        {
            lDir = normalize(eta * i - (eta * negNdotV + sqrtf(k)) * nn);
            return true;
        }
    }

    //Schlick approximation of Fresnel reflectance
    static __forceinline__  __device__
        float fresnel_schlick(const float cos_theta, const float exponent = 3.0f,
            const float minimum = 0.1f, const float maximum = 1.0f)
    {
        /*
          Clamp the result of the arithmetic due to floating point precision:
          the result should lie strictly within [minimum, maximum]
          return clamp(minimum + (maximum - minimum) * powf(1.0f - cos_theta, exponent),
                       minimum, maximum);
        */

        /* The max doesn'rDir seem like it should be necessary, but without it you get
            annoying broken pixels at the center of reflective spheres where cos_theta ~ 1.
        */
        return clamp(minimum + (maximum - minimum) * powf(fmaxf(0.0f, 1.0f - cos_theta), exponent),
            minimum, maximum);
    }


    static __forceinline__  __device__
        vec3f AxisAngle(const vec3f& w, const float cos2theta, const float phi)
    {
        const float cos_theta = std::sqrt(cos2theta);
        const float sin_theta = std::sqrt(1 - cos2theta);
        const vec3f u = normalize(cross(std::abs(w[0]) > float(.1) ? vec3f(0, 1, 0) : vec3f(1, 0, 0), w));
        const vec3f v = cross(w, u);
        return normalize(u * std::cos(phi) * sin_theta + v * std::sin(phi) * sin_theta + w * cos_theta);
    }
    
    static __forceinline__  __device__
        vec3f Sample(const vec3f diffuse, const vec3f spec, const float alpha, const vec3f& n, const vec3f& wi, vec3f& weight)
    {
        const float PI_ = 3.1415926535897932384626;
        const float k_d_ = (diffuse.x + diffuse.y + diffuse.z) / 3;
        const float k_s_ = (spec.x + spec.y + spec.z) / 3;
        const float R = k_s_ ? k_d_ / (k_d_ + k_s_) : 1.f;
        Random x;
        const float r0 = x();
        if (r0 < R) { // sample diffuse ray
            weight = k_d_ ? diffuse / R : vec3f(0, 0, 0);
            return AxisAngle(n, x(), x() * 2 * PI_);
        }

        else { // sample specular ray
            if (0) {
                const vec3f d = AxisAngle(n * 2 * dot(n, wi) - wi, std::pow(x(), float(2) / (alpha + 2)), x() * 2 * PI_);
                weight = dot(n, d) <= 0 || !k_s_ ? vec3f(0, 0, 0) : spec / (1 - R);
                return d;
            }
            else { // for ideal mirrors
                weight = k_s_ ? spec / (1 - R) : vec3f(0, 0, 0);
                return n * 2 * dot(n, wi) - wi;
            }
        }
    }

    // yqy's code below-------------------
        // tool functions---------------
    static __forceinline__  __device__
        float smithG_GGX(float NDotv, float alphaG)
    {
        float a = alphaG * alphaG;
        float b = NDotv * NDotv;
        return 1.0f / (NDotv + sqrtf(a + b - a * b));
    }
    static __forceinline__  __device__
        float SchlickFresnel(float u)
    {
        float m = clamp(1.0f - u, 0.0f, 1.0f);
        float m2 = m * m;
        return m2 * m2 * m; // pow(m,5)
    }
    static __forceinline__  __device__
        float GTR1(float NDotH, float a)
    {
        if (a >= 1.0f) return (1.0f / M_PI);
        float a2 = a * a;
        float t = 1.0f + (a2 - 1.0f) * NDotH * NDotH;
        return (a2 - 1.0f) / (M_PI * logf(a2) * t);
    }
    static __forceinline__  __device__
        float GTR2(float NDotH, float a)
    {
        float a2 = a * a;
        float t = 1.0f + (a2 - 1.0f) * NDotH * NDotH;
        return a2 / (M_PI * t * t);
    }
    template <typename T>
    static __forceinline__  __device__
        T lerp(T a, T b, float t)
    {
        return a+t*(b-a);
    }

    // PDF returns a percentage
    static __forceinline__  __device__
        float Pdf(const TriangleMeshSBTData& mat, const vec3f& normal, const vec3f& ray_in, const vec3f& ray_out)
    {
        vec3f n = normal;
        vec3f V = -ray_in;
        vec3f L = ray_out;

        float specularAlpha = max(0.001f, mat.roughness);
        //float clearcoatAlpha = std::lerp(0.1f, 0.001f, mat.clearcoatGloss);// 1.0 default
        float clearcoatAlpha = lerp(0.1f, 0.001f, 1.0f);
        
        float diffuseRatio = 0.5f * (1.f - mat.metallic);
        float specularRatio = 1.f - diffuseRatio;

        vec3f half = normalize(L + V);

        float cosTheta = abs(dot(half, n));
        float pdfGTR2 = GTR2(cosTheta, specularAlpha) * cosTheta;
        float pdfGTR1 = GTR1(cosTheta, clearcoatAlpha) * cosTheta;

        // calculate diffuse and specular pdfs and mix ratio
        //float ratio = 1.0f / (1.0f + mat.clearcoat);//0.0 default
        float ratio = 1.0f / (1.0f + 0.0f);
        float pdfSpec = lerp(pdfGTR1, pdfGTR2, ratio) / (4.0 * abs(dot(L, half)));
        float pdfDiff = abs(dot(L, n)) * (1.0f / M_PI);

        // weight pdfs according to ratios
        return diffuseRatio * pdfDiff + specularRatio * pdfSpec;
    }

    static __forceinline__  __device__
        vec3f Sample_yqy(const TriangleMeshSBTData& mat, const vec3f& normal, const vec3f& ray_in)
    {        
        vec3f N = normal;
        vec3f V = -ray_in;

        vec3f dir;
        Random x;
        //const float probability = x();
        float diffuseRatio = 0.5f * (1.0f - mat.metallic);

        //const float r1 = y();
        //const float r2 = z();
        const float r2 = x();

        if (x() < diffuseRatio) // sample diffuse
        {
            dir = AxisAngle(N, x(), x() * 2 * M_PI);
        }
        else
        {
            float a = max(0.001f, mat.roughness);

            float phi = x() * 2.0f * M_PI;

            float cosTheta = sqrtf((1.0f - r2) / (1.0f + (a * a - 1.0f) * r2));
            //float sinTheta = sqrtf(1.0f - (cosTheta * cosTheta));
            //float sinPhi = sinf(phi);
            //float cosPhi = cosf(phi);

            vec3f half = AxisAngle(N, 2*cosTheta*cosTheta-1, phi);

            dir = 2.0f * dot(V, half) * half - V; //reflection vector

        }
        return dir;
    }

    static __forceinline__  __device__
        vec3f Eval(const TriangleMeshSBTData& mat, const vec3f& normal, const vec3f& ray_in, const vec3f& ray_out)
    {
        vec3f N = normal;
        vec3f V = -ray_in;
        vec3f L = ray_out;

        float NDotL = dot(N, L);
        float NDotV = dot(N, V);
        if (NDotL <= 0.0f || NDotV <= 0.0f) return vec3f(0.0f);

        vec3f H = normalize(L + V);
        float NDotH = dot(N, H);
        float LDotH = dot(L, H);

        vec3f Cdlin = mat.color;

        float Cdlum = 0.3f * Cdlin.x + 0.6f * Cdlin.y + 0.1f * Cdlin.z; // luminance approx.

        vec3f Ctint = Cdlum > 0.0f ? Cdlin / Cdlum : vec3f(1.0f); // normalize lum. to isolate hue+sat
        //vec3f Cspec0 = lerp(mat.specular * 0.08f * lerp(vec3f(1.0f), Ctint, mat.specularTint), Cdlin, mat.metallic);
        //vec3f Csheen = lerp(vec3f(1.0f), Ctint, mat.sheenTint);
        vec3f Cspec0 = lerp(0.5f * 0.08f * lerp(vec3f(1.0f), Ctint, 0.0f), Cdlin, mat.metallic);
        vec3f Csheen = lerp(vec3f(1.0f), Ctint, 0.0f);

        // Diffuse fresnel - go from 1 at normal incidence to .5 at grazing
        // and mix in diffuse retro-reflection based on roughness
        float FL = SchlickFresnel(NDotL), FV = SchlickFresnel(NDotV);
        float Fd90 = 0.5f + 2.0f * LDotH * LDotH * mat.roughness;
        float Fd = lerp(1.0f, Fd90, FL) * lerp(1.0f, Fd90, FV);

        // Based on Hanrahan-Krueger brdf approximation of isotrokPic bssrdf
        // 1.25 scale is used to (roughly) preserve albedo
        // Fss90 used to "flatten" retroreflection based on roughness
        float Fss90 = LDotH * LDotH * mat.roughness;
        float Fss = lerp(1.0f, Fss90, FL) * lerp(1.0f, Fss90, FV);
        float ss = 1.25f * (Fss * (1.0f / (NDotL + NDotV) - 0.5f) + 0.5f);

        // specular
        //float aspect = sqrt(1-mat.anisotrokPic*.9);
        //float ax = Max(.001f, sqr(mat.roughness)/aspect);
        //float ay = Max(.001f, sqr(mat.roughness)*aspect);
        //float Ds = GTR2_aniso(NDotH, Dot(H, X), Dot(H, Y), ax, ay);

        float a = max(0.001f, mat.roughness);
        float Ds = GTR2(NDotH, a);
        float FH = SchlickFresnel(LDotH);
        vec3f Fs = lerp(Cspec0, vec3f(1.0f), FH);
        float roughg = sqrt(mat.roughness * 0.5f + 0.5f);
        float Gs = smithG_GGX(NDotL, roughg) * smithG_GGX(NDotV, roughg);

        // sheen
        vec3f Fsheen = FH * mat.sheen * Csheen;

        // clearcoat (ior = 1.5 -> F0 = 0.04)
        //float Dr = GTR1(NDotH, lerp(0.1f, 0.001f, mat.clearcoatGloss));
        float Dr = GTR1(NDotH, lerp(0.1f, 0.001f, 1.0f));
        float Fr = lerp(0.04f, 1.0f, FH);
        float Gr = smithG_GGX(NDotL, 0.25f) * smithG_GGX(NDotV, 0.25f);

        //vec3f out = ((1.0f / M_PI) * lerp(Fd, ss, mat.subsurface) * Cdlin + Fsheen)
        //    * (1.0f - mat.metallic)
         //   + Gs * Fs * Ds + 0.25f * mat.clearcoat * Gr * Fr * Dr;
        vec3f out = ((1.0f / M_PI) * lerp(Fd, ss, 0.0f) * Cdlin + Fsheen)
            * (1.0f - mat.metallic)
            + Gs * Fs * Ds + 0.25f * 0.0f * Gr * Fr * Dr;
        return out * clamp(dot(N, L), 0.0f, 1.0f);
    }
    //------------------------------------------------------------------------------
    // closest hit and anyhit programs for radiance-type rays.
    //
    // Note eventually we will have to create one pair of those for each
    // ray type and each geometry type we want to render; but this
    // simple example doesn't use any actual geometries yet, so we only
    // create a single, dummy, set of them (we do have to have at least
    // one group of them to set up the SBT)
    //------------------------------------------------------------------------------

    extern "C" __global__ void __closesthit__shadow()
    {
        /* not going to be used ... */
    }

    extern "C" __global__ void __closesthit__radiance()
    {
        const TriangleMeshSBTData& sbtData
            = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();
        PRD& prd = *getPRD<PRD>();
        const int Maxdepth = 4;
        const float refraction_color = 1.0f;
        const float reflection_color = 1.0f;
        if (prd.depth >= Maxdepth) {
            prd.pixelColor = 0.0f;
            return;
        }
        if (sbtData.emissive_) {
            prd.pixelColor *= sbtData.emission;
            return;
        }
        // ------------------------------------------------------------------
        // gather some basic hit information
        // ------------------------------------------------------------------
        const int   primID = optixGetPrimitiveIndex();
        const vec3i index = sbtData.index[primID];
        const float u = optixGetTriangleBarycentrics().x;
        const float v = optixGetTriangleBarycentrics().y;

        // ------------------------------------------------------------------
        // compute normal, using either shading normal (if avail), or
        // geometry normal (fallback)
        // ------------------------------------------------------------------
        const vec3f& A = sbtData.vertex[index.x];
        const vec3f& B = sbtData.vertex[index.y];
        const vec3f& C = sbtData.vertex[index.z];
        vec3f Ng = cross(B - A, C - A);
        vec3f Ns = (sbtData.normal)
            ? ((1.f - u - v) * sbtData.normal[index.x]
                + u * sbtData.normal[index.y]
                + v * sbtData.normal[index.z])
            : Ng;

        // ------------------------------------------------------------------
        // face-forward and normalize normals
        // ------------------------------------------------------------------
        const vec3f rayDir = optixGetWorldRayDirection();

        if (dot(rayDir, Ng) > 0.f) Ng = -Ng;
        Ng = normalize(Ng);

        if (dot(Ng, Ns) < 0.f)
            Ns -= 2.f * dot(Ng, Ns) * Ng;
        Ns = normalize(Ns);

        // ------------------------------------------------------------------
        // compute diffuse material color, including diffuse texture, if
        // available
        // ------------------------------------------------------------------
        vec3f diffuseColor = sbtData.color;
        if (sbtData.hasTexture && sbtData.texcoord) {
            const vec2f tc
                = (1.f - u - v) * sbtData.texcoord[index.x]
                + u * sbtData.texcoord[index.y]
                + v * sbtData.texcoord[index.z];

            vec4f fromTexture = tex2D<float4>(sbtData.texture, tc.x, tc.y);
            diffuseColor *= (vec3f)fromTexture;
        }

        vec3f specColor = 0.0f;
        if (sbtData.hasSpecTexture && sbtData.texcoord) {
            const vec2f tc
                = (1.f - u - v) * sbtData.texcoord[index.x]
                + u * sbtData.texcoord[index.y]
                + v * sbtData.texcoord[index.z];
            vec4f fromTexture = tex2D<float4>(sbtData.spectexture, tc.x, tc.y);
            specColor = (vec3f)fromTexture;
        }

        const float alpha = sbtData.alpha_;
        const float d = sbtData.d;

        // start with some ambient term
        //vec3f pixelColor = (0.1f + 0.2f*fabsf(dot(Ns,rayDir)))*diffuseColor;
        vec3f pixelColor = 0.f;

        // ------------------------------------------------------------------
        // compute shadow
        // ------------------------------------------------------------------
        const vec3f surfPos
            = (1.f - u - v) * sbtData.vertex[index.x]
            + u * sbtData.vertex[index.y]
            + v * sbtData.vertex[index.z];

        const int numLightSamples = NUM_LIGHT_SAMPLES;
        for (int lightSampleID = 0; lightSampleID < numLightSamples; lightSampleID++) {
            float reflection = 1.0f;
            vec3f rDir;//����
            float cos_theta = dot(rayDir, Ns);
            if (d < 0.5 && refract(rDir, rayDir, Ns, prd.refraction_index))
            {
                //��������ǵ�cosֵ�����������뷴��ı��ʣ���һ���ǶȾ�ȫ�����ˣ��ڴ�ֱʱ��ȫ����
                //�������ȥ,�����������
                if (cos_theta < 0.0f)
                {
                    cos_theta = -cos_theta;
                }
                else
                {
                    //�쳣��ʹ����������ټ���һ��
                    cos_theta = dot(rDir, Ns);
                }

                reflection = fresnel_schlick(cos_theta);
                float rimportance =  (1.0f - reflection) * refraction_color;
                PRD newprd;
                // the values we store the PRD pointer in:
                uint32_t u0, u1;
                packPointer(&newprd, u0, u1);
                newprd.pixelColor = prd.pixelColor * rimportance;
                newprd.depth = prd.depth + 1;
                if (prd.refraction_index > 1.f)
                    newprd.refraction_index = 0.684f;
                else
                    newprd.refraction_index = 1.46f;
                optixTrace(optixLaunchParams.traversable,
                    surfPos - 1e-3f * Ng,
                    rDir,
                    0.f,    // tmin
                    1e20f,  // tmax
                    0.0f,   // rayTime
                    OptixVisibilityMask(255),
                    // For shadow rays: skip any/closest hit shaders and terminate on first
                    // intersection with anything. The miss shader is used to mark if the
                    // light was visible.
                    OPTIX_RAY_FLAG_DISABLE_ANYHIT,
                    RADIANCE_RAY_TYPE,            // SBT offset
                    RAY_TYPE_COUNT,               // SBT stride
                    RADIANCE_RAY_TYPE,            // missSBTIndex 
                    u0, u1);
                pixelColor +=   newprd.pixelColor / numLightSamples;
            }
            if (cos_theta < 0.0f) {
                float limportance = reflection * reflection_color;
                // the values we store the PRD pointer in:
                PRD newprd;
                vec3f weight = 1.0f;
                vec3f mont_dir = Sample(diffuseColor, specColor, alpha, Ns, -rayDir, weight);
                //vec3f mont_dir = Sample_yqy(sbtData, Ns, rayDir);
                //weight = Eval(sbtData, Ns, rayDir, mont_dir);
                uint32_t u0, u1;
                packPointer(&newprd, u0, u1);
                newprd.depth = prd.depth + 1;
                newprd.refraction_index = 1.0;
                newprd.pixelColor = prd.pixelColor  * weight* limportance;
                optixTrace(optixLaunchParams.traversable,
                    surfPos + 1e-3f * Ng,
                    mont_dir,
                    0.f,    // tmin
                    1e20f,  // tmax
                    0.0f,   // rayTime
                    OptixVisibilityMask(255),
                    // For shadow rays: skip any/closest hit shaders and terminate on first
                    // intersection with anything. The miss shader is used to mark if the
                    // light was visible.
                    OPTIX_RAY_FLAG_DISABLE_ANYHIT,
                    RADIANCE_RAY_TYPE,            // SBT offset
                    RAY_TYPE_COUNT,               // SBT stride
                    RADIANCE_RAY_TYPE,            // missSBTIndex 
                    u0, u1);
                pixelColor +=  newprd.pixelColor / numLightSamples;
            }
        }
        prd.pixelNormal = Ns;
        prd.pixelAlbedo = diffuseColor;
        prd.pixelColor = pixelColor;
    }

    extern "C" __global__ void __anyhit__radiance()
    { /*! for this simple example, this will remain empty */
    }

    extern "C" __global__ void __anyhit__shadow()
    { /*! not going to be used */
    }

    //------------------------------------------------------------------------------
    // miss program that gets called for any ray that did not have a
    // valid intersection
    //
    // as with the anyhit/closest hit programs, in this example we only
    // need to have _some_ dummy function to set up a valid SBT
    // ------------------------------------------------------------------------------

    extern "C" __global__ void __miss__radiance()
    {
        PRD& prd = *getPRD<PRD>();
        // set to constant white as background color
        prd.pixelColor *= vec3f(10.f);
    }

    extern "C" __global__ void __miss__shadow()
    {
        // we didn't hit anything, so the light is visible
        vec3f& prd = *(vec3f*)getPRD<vec3f>();
        prd = vec3f(0.f);
    }

    //------------------------------------------------------------------------------
    // ray gen program - the actual rendering happens in here
    //------------------------------------------------------------------------------
    extern "C" __global__ void __raygen__renderFrame()
    {
        // compute a test pattern based on pixel ID
        const int ix = optixGetLaunchIndex().x;
        const int iy = optixGetLaunchIndex().y;
        const auto& camera = optixLaunchParams.camera;

        PRD prd;
        prd.random.init(ix + optixLaunchParams.frame.size.x * iy,
            optixLaunchParams.frame.frameID);

        // the values we store the PRD pointer in:
        uint32_t u0, u1;
        packPointer(&prd, u0, u1);

        int numPixelSamples = optixLaunchParams.numPixelSamples;

        vec3f pixelColor = 0.f;
        vec3f pixelNormal = 0.f;
        vec3f pixelAlbedo = 0.f;
        for (int sampleID = 0; sampleID < numPixelSamples; sampleID++) {
            // normalized screen plane position, in [0,1]^2

            // iw: note for denoising that's not actually correct - if we
            // assume that the camera should only(!) cover the denoised
            // screen then the actual screen plane we shuld be using during
            // rendreing is slightly larger than [0,1]^2
            vec2f screen(vec2f(ix + prd.random(), iy + prd.random())
                / vec2f(optixLaunchParams.frame.size));
            // screen
            //   = screen
            //   * vec2f(optixLaunchParams.frame.denoisedSize)
            //   * vec2f(optixLaunchParams.frame.size)
            //   - 0.5f*(vec2f(optixLaunchParams.frame.size)
            //           -
            //           vec2f(optixLaunchParams.frame.denoisedSize)
            //           );

            // generate ray direction
            vec3f rayDir = normalize(camera.direction
                + (screen.x - 0.5f) * camera.horizontal
                + (screen.y - 0.5f) * camera.vertical);

            prd.pixelColor = vec3f(1.f);
            prd.depth = 0;
            prd.refraction_index = 1.46f;
            optixTrace(optixLaunchParams.traversable,
                camera.position,
                rayDir,
                0.f,    // tmin
                1e20f,  // tmax
                0.0f,   // rayTime
                OptixVisibilityMask(255),
                OPTIX_RAY_FLAG_DISABLE_ANYHIT,//OPTIX_RAY_FLAG_NONE,
                RADIANCE_RAY_TYPE,            // SBT offset
                RAY_TYPE_COUNT,               // SBT stride
                RADIANCE_RAY_TYPE,            // missSBTIndex 
                u0, u1);
            pixelColor += prd.pixelColor;
            pixelNormal += prd.pixelNormal;
            pixelAlbedo += prd.pixelAlbedo;
        }

        vec4f rgba(pixelColor / numPixelSamples, 1.f);
        vec4f albedo(pixelAlbedo / numPixelSamples, 1.f);
        vec4f normal(pixelNormal / numPixelSamples, 1.f);

        // and write/accumulate to frame buffer ...
        const uint32_t fbIndex = ix + iy * optixLaunchParams.frame.size.x;
        if (optixLaunchParams.frame.frameID > 0) {
            rgba
                += float(optixLaunchParams.frame.frameID)
                * vec4f(optixLaunchParams.frame.colorBuffer[fbIndex]);
            rgba /= (optixLaunchParams.frame.frameID + 1.f);
        }
        optixLaunchParams.frame.colorBuffer[fbIndex] = (float4)rgba;
        optixLaunchParams.frame.albedoBuffer[fbIndex] = (float4)albedo;
        optixLaunchParams.frame.normalBuffer[fbIndex] = (float4)normal;
    }

} // ::osc
