#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2018-2019 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include <optix_device.h>
#include <hip/hip_runtime.h>

#include "LaunchParams.h"
#include "PRD.h"

using namespace osc;

#define NUM_LIGHT_SAMPLES 2

namespace osc {


    /*! launch parameters in constant memory, filled in by optix upon
        optixLaunch (this gets filled in from the buffer we pass to
        optixLaunch) */
    extern "C" __constant__ LaunchParams optixLaunchParams;

    static __forceinline__ __device__
        void* unpackPointer(uint32_t i0, uint32_t i1)
    {
        const uint64_t uptr = static_cast<uint64_t>(i0) << 32 | i1;
        void* ptr = reinterpret_cast<void*>(uptr);
        return ptr;
    }

    static __forceinline__ __device__
        void  packPointer(void* ptr, uint32_t& i0, uint32_t& i1)
    {
        const uint64_t uptr = reinterpret_cast<uint64_t>(ptr);
        i0 = uptr >> 32;
        i1 = uptr & 0x00000000ffffffff;
    }

    template<typename T>
    static __forceinline__ __device__ T* getPRD()
    {
        const uint32_t u0 = optixGetPayload_0();
        const uint32_t u1 = optixGetPayload_1();
        return reinterpret_cast<T*>(unpackPointer(u0, u1));
    }

    /*
*  Calculates refraction direction
*  lDir   : refraction vector
*  i   : incident vector
*  n   : surface normal
*  ior : index of refraction ( n2 / n1 )
*  returns false in case of total internal reflection, in that case lDir is
*  initialized to (0,0,0).
*/
    static __forceinline__  __device__
        bool refract(vec3f& lDir, vec3f const& i, vec3f const& n, const float ior)
    {
        vec3f nn = n;
        float negNdotV = dot(i, nn);
        float eta;

        if (negNdotV > 0.0f)
        {
            eta = ior;
            nn = -n;
            negNdotV = -negNdotV;
        }
        else
        {
            eta = 1.f / ior;
        }

        const float k = 1.f - eta * eta * (1.f - negNdotV * negNdotV);

        if (k < 0.0f)
        {
            // Initialize this value, so that lDir always leaves this function initialized.
            lDir = vec3f(0.f);
            return false;
        }
        else
        {
            lDir = normalize(eta * i - (eta * negNdotV + sqrtf(k)) * nn);
            return true;
        }
    }

    //Schlick approximation of Fresnel reflectance
    static __forceinline__  __device__
        float fresnel_schlick(const float cos_theta, const float exponent = 3.0f,
            const float minimum = 0.1f, const float maximum = 1.0f)
    {
        /*
          Clamp the result of the arithmetic due to floating point precision:
          the result should lie strictly within [minimum, maximum]
          return clamp(minimum + (maximum - minimum) * powf(1.0f - cos_theta, exponent),
                       minimum, maximum);
        */

        /* The max doesn'rDir seem like it should be necessary, but without it you get
            annoying broken pixels at the center of reflective spheres where cos_theta ~ 1.
        */
        return clamp(minimum + (maximum - minimum) * powf(fmaxf(0.0f, 1.0f - cos_theta), exponent),
            minimum, maximum);
    }


    static __forceinline__  __device__
        vec3f AxisAngle(const vec3f& w, const float cos2theta, const float phi)
    {
        const float cos_theta = std::sqrt(cos2theta);
        const float sin_theta = std::sqrt(1 - cos2theta);
        const vec3f u = normalize(cross(std::abs(w[0]) > float(.1) ? vec3f(0, 1, 0) : vec3f(1, 0, 0), w));
        const vec3f v = cross(w, u);
        return normalize(u * std::cos(phi) * sin_theta + v * std::sin(phi) * sin_theta + w * cos_theta);
    }

    static __forceinline__  __device__
        vec3f Sample(const vec3f diffuse, const vec3f spec, const float alpha, const vec3f& n, const vec3f& wi, vec3f& weight)
    {
        const float PI_ = 3.1415926535897932384626;
        const float k_d_ = (diffuse.x + diffuse.y + diffuse.z) / 3;
        const float k_s_ = (spec.x + spec.y + spec.z) / 3;
        const float R = k_s_ ? k_d_ / (k_d_ + k_s_) : 1.f;
        Random x;
        const float r0 = x();
        if (r0 < R) { // sample diffuse ray
            weight = k_d_ ? diffuse / R : vec3f(0, 0, 0);
            return AxisAngle(n, x(), x() * 2 * PI_);
        }

        else { // sample specular ray
            if (0) {
                const vec3f d = AxisAngle(n * 2 * dot(n, wi) - wi, std::pow(x(), float(2) / (alpha + 2)), x() * 2 * PI_);
                weight = dot(n, d) <= 0 || !k_s_ ? vec3f(0, 0, 0) : spec / (1 - R);
                return d;
            }
            else { // for ideal mirrors
                weight = k_s_ ? spec / (1 - R) : vec3f(0, 0, 0);
                return n * 2 * dot(n, wi) - wi;
            }
        }
    }

    //------------------------------------------------------------------------------
    // closest hit and anyhit programs for radiance-type rays.
    //
    // Note eventually we will have to create one pair of those for each
    // ray type and each geometry type we want to render; but this
    // simple example doesn't use any actual geometries yet, so we only
    // create a single, dummy, set of them (we do have to have at least
    // one group of them to set up the SBT)
    //------------------------------------------------------------------------------

    extern "C" __global__ void __closesthit__shadow()
    {
        /* not going to be used ... */
    }

    extern "C" __global__ void __closesthit__radiance()
    {
        const TriangleMeshSBTData& sbtData
            = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();
        PRD& prd = *getPRD<PRD>();
        const int Maxdepth = 4;
        const float refraction_color = 1.0f;
        const float reflection_color = 1.0f;
        if (prd.depth >= Maxdepth) {
            prd.pixelColor = 0.0f;
            return;
        }
        if (sbtData.emissive_) {
            prd.pixelColor *= sbtData.emission;
            return;
        }
        // ------------------------------------------------------------------
        // gather some basic hit information
        // ------------------------------------------------------------------
        const int   primID = optixGetPrimitiveIndex();
        const vec3i index = sbtData.index[primID];
        const float u = optixGetTriangleBarycentrics().x;
        const float v = optixGetTriangleBarycentrics().y;

        // ------------------------------------------------------------------
        // compute normal, using either shading normal (if avail), or
        // geometry normal (fallback)
        // ------------------------------------------------------------------
        const vec3f& A = sbtData.vertex[index.x];
        const vec3f& B = sbtData.vertex[index.y];
        const vec3f& C = sbtData.vertex[index.z];
        vec3f Ng = cross(B - A, C - A);
        vec3f Ns = (sbtData.normal)
            ? ((1.f - u - v) * sbtData.normal[index.x]
                + u * sbtData.normal[index.y]
                + v * sbtData.normal[index.z])
            : Ng;

        // ------------------------------------------------------------------
        // face-forward and normalize normals
        // ------------------------------------------------------------------
        const vec3f rayDir = optixGetWorldRayDirection();

        if (dot(rayDir, Ng) > 0.f) Ng = -Ng;
        Ng = normalize(Ng);

        if (dot(Ng, Ns) < 0.f)
            Ns -= 2.f * dot(Ng, Ns) * Ng;
        Ns = normalize(Ns);

        // ------------------------------------------------------------------
        // compute diffuse material color, including diffuse texture, if
        // available
        // ------------------------------------------------------------------
        vec3f diffuseColor = sbtData.color;
        if (sbtData.hasTexture && sbtData.texcoord) {
            const vec2f tc
                = (1.f - u - v) * sbtData.texcoord[index.x]
                + u * sbtData.texcoord[index.y]
                + v * sbtData.texcoord[index.z];

            vec4f fromTexture = tex2D<float4>(sbtData.texture, tc.x, tc.y);
            diffuseColor *= (vec3f)fromTexture;
        }

        vec3f specColor = 0.0f;
        if (sbtData.hasSpecTexture && sbtData.texcoord) {
            const vec2f tc
                = (1.f - u - v) * sbtData.texcoord[index.x]
                + u * sbtData.texcoord[index.y]
                + v * sbtData.texcoord[index.z];
            vec4f fromTexture = tex2D<float4>(sbtData.spectexture, tc.x, tc.y);
            specColor = (vec3f)fromTexture;
        }

        const float alpha = sbtData.alpha_;
        const float d = sbtData.d;

        // start with some ambient term
        //vec3f pixelColor = (0.1f + 0.2f*fabsf(dot(Ns,rayDir)))*diffuseColor;
        vec3f pixelColor = 0.f;

        // ------------------------------------------------------------------
        // compute shadow
        // ------------------------------------------------------------------
        const vec3f surfPos
            = (1.f - u - v) * sbtData.vertex[index.x]
            + u * sbtData.vertex[index.y]
            + v * sbtData.vertex[index.z];

        const int numLightSamples = NUM_LIGHT_SAMPLES;
        for (int lightSampleID = 0; lightSampleID < numLightSamples; lightSampleID++) {
            float reflection = 1.0f;
            vec3f rDir;//����
            float cos_theta = dot(rayDir, Ns);
            if (d < 0.5 && refract(rDir, rayDir, Ns, prd.refraction_index))
            {
                //��������ǵ�cosֵ�����������뷴��ı��ʣ���һ���ǶȾ�ȫ�����ˣ��ڴ�ֱʱ��ȫ����
                //�������ȥ,�����������
                if (cos_theta < 0.0f)
                {
                    cos_theta = -cos_theta;
                }
                else
                {
                    //�쳣��ʹ����������ټ���һ��
                    cos_theta = dot(rDir, Ns);
                }

                reflection = fresnel_schlick(cos_theta);
                float rimportance =  (1.0f - reflection) * refraction_color;
                PRD newprd;
                // the values we store the PRD pointer in:
                uint32_t u0, u1;
                packPointer(&newprd, u0, u1);
                newprd.pixelColor = prd.pixelColor * rimportance;
                newprd.depth = prd.depth + 1;
                if (prd.refraction_index > 1.f)
                    newprd.refraction_index = 0.684f;
                else
                    newprd.refraction_index = 1.46f;
                optixTrace(optixLaunchParams.traversable,
                    surfPos - 1e-3f * Ng,
                    rDir,
                    0.f,    // tmin
                    1e20f,  // tmax
                    0.0f,   // rayTime
                    OptixVisibilityMask(255),
                    // For shadow rays: skip any/closest hit shaders and terminate on first
                    // intersection with anything. The miss shader is used to mark if the
                    // light was visible.
                    OPTIX_RAY_FLAG_DISABLE_ANYHIT,
                    RADIANCE_RAY_TYPE,            // SBT offset
                    RAY_TYPE_COUNT,               // SBT stride
                    RADIANCE_RAY_TYPE,            // missSBTIndex 
                    u0, u1);
                pixelColor +=   newprd.pixelColor / numLightSamples;
            }
            if (cos_theta < 0.0f) {
                float limportance = reflection * reflection_color;
                // the values we store the PRD pointer in:
                PRD newprd;
                vec3f weight = 1.0f;
                vec3f mont_dir = Sample(diffuseColor, specColor, alpha, Ns, -rayDir, weight);
                uint32_t u0, u1;
                packPointer(&newprd, u0, u1);
                newprd.depth = prd.depth + 1;
                newprd.refraction_index = 1.0;
                newprd.pixelColor = prd.pixelColor  * weight* limportance;
                optixTrace(optixLaunchParams.traversable,
                    surfPos + 1e-3f * Ng,
                    mont_dir,
                    0.f,    // tmin
                    1e20f,  // tmax
                    0.0f,   // rayTime
                    OptixVisibilityMask(255),
                    // For shadow rays: skip any/closest hit shaders and terminate on first
                    // intersection with anything. The miss shader is used to mark if the
                    // light was visible.
                    OPTIX_RAY_FLAG_DISABLE_ANYHIT,
                    RADIANCE_RAY_TYPE,            // SBT offset
                    RAY_TYPE_COUNT,               // SBT stride
                    RADIANCE_RAY_TYPE,            // missSBTIndex 
                    u0, u1);
                pixelColor +=  newprd.pixelColor / numLightSamples;
            }
        }
        prd.pixelNormal = Ns;
        prd.pixelAlbedo = diffuseColor;
        prd.pixelColor = pixelColor;
    }

    extern "C" __global__ void __anyhit__radiance()
    { /*! for this simple example, this will remain empty */
    }

    extern "C" __global__ void __anyhit__shadow()
    { /*! not going to be used */
    }

    //------------------------------------------------------------------------------
    // miss program that gets called for any ray that did not have a
    // valid intersection
    //
    // as with the anyhit/closest hit programs, in this example we only
    // need to have _some_ dummy function to set up a valid SBT
    // ------------------------------------------------------------------------------

    extern "C" __global__ void __miss__radiance()
    {
        PRD& prd = *getPRD<PRD>();
        // set to constant white as background color
        // �������Ҫ�ĳɻ�����
        prd.pixelColor *= vec3f(10.f);
    }

    extern "C" __global__ void __miss__shadow()
    {
        // we didn't hit anything, so the light is visible
        vec3f& prd = *(vec3f*)getPRD<vec3f>();
        prd = vec3f(0.f);
    }

    //------------------------------------------------------------------------------
    // ray gen program - the actual rendering happens in here
    //------------------------------------------------------------------------------
    extern "C" __global__ void __raygen__renderFrame()
    {
        // compute a test pattern based on pixel ID
        const int ix = optixGetLaunchIndex().x;
        const int iy = optixGetLaunchIndex().y;
        const auto& camera = optixLaunchParams.camera;

        PRD prd;
        prd.random.init(ix + optixLaunchParams.frame.size.x * iy,
            optixLaunchParams.frame.frameID);

        // the values we store the PRD pointer in:
        uint32_t u0, u1;
        packPointer(&prd, u0, u1);

        int numPixelSamples = optixLaunchParams.numPixelSamples;

        vec3f pixelColor = 0.f;
        vec3f pixelNormal = 0.f;
        vec3f pixelAlbedo = 0.f;
        for (int sampleID = 0; sampleID < numPixelSamples; sampleID++) {
            // normalized screen plane position, in [0,1]^2

            // iw: note for denoising that's not actually correct - if we
            // assume that the camera should only(!) cover the denoised
            // screen then the actual screen plane we shuld be using during
            // rendreing is slightly larger than [0,1]^2
            vec2f screen(vec2f(ix + prd.random(), iy + prd.random())
                / vec2f(optixLaunchParams.frame.size));
            // screen
            //   = screen
            //   * vec2f(optixLaunchParams.frame.denoisedSize)
            //   * vec2f(optixLaunchParams.frame.size)
            //   - 0.5f*(vec2f(optixLaunchParams.frame.size)
            //           -
            //           vec2f(optixLaunchParams.frame.denoisedSize)
            //           );

            // generate ray direction
            vec3f rayDir = normalize(camera.direction
                + (screen.x - 0.5f) * camera.horizontal
                + (screen.y - 0.5f) * camera.vertical);

            prd.pixelColor = vec3f(1.f);
            prd.depth = 0;
            prd.refraction_index = 1.46f;
            optixTrace(optixLaunchParams.traversable,
                camera.position,
                rayDir,
                0.f,    // tmin
                1e20f,  // tmax
                0.0f,   // rayTime
                OptixVisibilityMask(255),
                OPTIX_RAY_FLAG_DISABLE_ANYHIT,//OPTIX_RAY_FLAG_NONE,
                RADIANCE_RAY_TYPE,            // SBT offset
                RAY_TYPE_COUNT,               // SBT stride
                RADIANCE_RAY_TYPE,            // missSBTIndex 
                u0, u1);
            pixelColor += prd.pixelColor;
            pixelNormal += prd.pixelNormal;
            pixelAlbedo += prd.pixelAlbedo;
        }

        vec4f rgba(pixelColor / numPixelSamples, 1.f);
        vec4f albedo(pixelAlbedo / numPixelSamples, 1.f);
        vec4f normal(pixelNormal / numPixelSamples, 1.f);

        // and write/accumulate to frame buffer ...
        const uint32_t fbIndex = ix + iy * optixLaunchParams.frame.size.x;
        if (optixLaunchParams.frame.frameID > 0) {
            rgba
                += float(optixLaunchParams.frame.frameID)
                * vec4f(optixLaunchParams.frame.colorBuffer[fbIndex]);
            rgba /= (optixLaunchParams.frame.frameID + 1.f);
        }
        optixLaunchParams.frame.colorBuffer[fbIndex] = (float4)rgba;
        optixLaunchParams.frame.albedoBuffer[fbIndex] = (float4)albedo;
        optixLaunchParams.frame.normalBuffer[fbIndex] = (float4)normal;
    }

} // ::osc
